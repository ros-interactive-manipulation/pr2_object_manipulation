#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, Mårten Björkman (celle@csc.kth.se) 
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *
 *  1.Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  2.Redistributions in binary form must reproduce the above
 *    copyright notice, this list of conditions and the following
 *    disclaimer in the documentation and/or other materials provided
 *    with the distribution.  
 *  3.The name of Mårten Björkman may not be used to endorse or
 *    promote products derived from this software without specific
 *    prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "cudautils.h"
#include "fgbgsegment.h"
#include "cudasegment.h"
#include "cudabelieft.h"
#include "timercpu.h"
#include "matrix3.h"

#define HSV_W 32 // RGBToHSV  
#define HSV_H 8
#define PRI_W 16 // ComputePriors
#define PRI_H 16 
#define BTP_W 32 // BeliefsToProbs
#define BTP_H 16 
#define SUG_W 128 // SubUpGradients
#define SUG_H 2
#define SGC_W 32  // SetGradientCosts
#define SGC_H 8

//#undef __SSE2__

static const int hist_size = FgBgSegment::hist_size;
static const int max_figures = CudaSegment::max_figures;
 
__constant__ float fispread3d_d[max_figures*9];
__constant__ float fposition3d_d[max_figures*3];
__constant__ float sconstants_d[4];
__constant__ float fcolcost_d[max_figures*hist_size*hist_size];
__constant__ float scolcost_d[hist_size*hist_size];
__constant__ float gcolcost_d[hist_size*hist_size];
__constant__ float constd_d[max_figures+2];
__constant__ float constp_d[max_figures+2];
__constant__ float const0_d[max_figures+2];
__constant__ float constu_d[max_figures+2]; 

__global__ void RGBToHSV(uchar *srcd, uint *bins, uchar *vimd, int width, int height) 
{  
  __shared__ uchar rgb[3*HSV_W*HSV_H];
  int lx = threadIdx.x;
  int ly = threadIdx.y;
  int bx = blockIdx.x*blockDim.x;
  int y = blockIdx.y*blockDim.y + ly;
  int bw = __mul24(y, width) + bx;
  uchar *crgb = &rgb[3*__mul24(ly, HSV_W)];
  int *irgb = (int *)crgb;
  int *isrc = (int *)&srcd[3*bw];
  int minw = 3*min(HSV_W, width - bx); 
  if (4*lx<minw && y<height)   
    irgb[lx] = isrc[lx]; 
  __syncthreads();
  int x = bx + lx;
  if (x<width && y<height) {
    int p = bw + lx;
    int r = crgb[3*lx+0];
    int g = crgb[3*lx+1];
    int b = crgb[3*lx+2];  
    int minv = min(r, min(g, b));
    int maxv = max(r, max(g, b));
    vimd[p] = maxv;
    int diff = maxv - minv;
    int dif6 = diff*6;
    int hue = 0, sat = 0;
    if (diff>0) {
      if (maxv==r) 
	hue = (1536*diff + 256*(g - b))/dif6 & 255;
      else if (maxv==g)
	hue =  (512*diff + 256*(b - r))/dif6;
      else 
	hue = (1024*diff + 256*(r - g))/dif6;
    }
    if (maxv>0)
      sat = 255*(maxv - minv)/maxv;
    int ix = hist_size*hue/256;
    int iy = hist_size*sat/256;
    int idx = iy*hist_size + ix;
    bins[p] = idx;
  }
}

__global__ void ComputePriors(uint *bins, float *dimd, int drange, float *gpriors, float *spriors, float *fpriors, int numFigures, int width, int height)
{
  int lx = threadIdx.x;
  int ly = threadIdx.y;
  int x = blockIdx.x*blockDim.x + lx;
  int y = blockIdx.y*blockDim.y + ly;
  if (x<width && y<height) {
    int p = y*width + x;
    float d = dimd[p];
    bool ok = (d>=0 && d<drange);
    int idx = bins[p];
    int sz = width*height;
    for (int f=0;f<numFigures;f++) {
      float er_f = const0_d[f+2];
      float difx = x - fposition3d_d[f*3+0];
      float dify = y - fposition3d_d[f*3+1];
      float difp = difx*(difx*fispread3d_d[f*9+0] + 2.0f*dify*fispread3d_d[f*9+1]) + dify*dify*fispread3d_d[f*9+4];
      er_f += constp_d[f+2] + (difp<25.0f ? difp : 100.0f);
      if (ok) {
	float difd = d - fposition3d_d[f*3+2];
	er_f += difd*(2.0f*difx*fispread3d_d[f*9+2] + 2.0f*dify*fispread3d_d[f*9+5] + difd*fispread3d_d[f*9+8]) + constd_d[f+2];
      } else 
	er_f += constu_d[f+2];
      er_f += fcolcost_d[f*hist_size*hist_size+idx];
      fpriors[f*sz+p] = 0.5f*er_f;
    }
    float er_g = const0_d[0];
    float er_s = const0_d[1];
    er_g += constp_d[0]; 
    er_s += constp_d[1];
    if (ok) {
      er_g += constd_d[0];
      float diff = d - (sconstants_d[0]*x + sconstants_d[1]*y + sconstants_d[2]);
      er_s += diff*diff*sconstants_d[3] + constd_d[1];
    } else {
      er_g += constu_d[0];
      er_s += constu_d[1];
    }
    er_g += gcolcost_d[idx];
    er_s += scolcost_d[idx]; 
    gpriors[p] = 0.5f*er_g;
    spriors[p] = 0.5f*er_s;
  }
}

__global__ void BeliefsToProbs(float *gbeliefs, float *sbeliefs, float *fbeliefs, float *gprobs, float *sprobs, float *fprobs, int numFigures, int width, int height)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int sz = width*height;
  float fb[max_figures];
  if (x<width && y<height) {
    int p = y*width + x;
    float gb = gbeliefs[p];
    float sb = sbeliefs[p];
    float minbelief = fminf(gb, sb);
    for (int f=0;f<numFigures;f++) {
      fb[f] = fbeliefs[f*sz+p];
      minbelief = fminf(minbelief, fb[f]);
    }
    float prob_g = __expf(minbelief - gb) + 1e-6f;
    float prob_s = __expf(minbelief - sb) + 1e-6f;
    float sumprob = prob_g + prob_s;
    for (int f=0;f<numFigures;f++) {
      fb[f] = __expf(minbelief - fb[f]) + 1e-6f;
      sumprob += fb[f];
    }
    float isumprob = __frcp_rn(sumprob);
    gprobs[p] = prob_g*isumprob;
    sprobs[p] = prob_s*isumprob;
    for (int f=0;f<numFigures;f++) 
      fprobs[f*sz+p] = fb[f]*isumprob;
  }
}

#if 0  // Only possible if histograms are small

#define HIST_W 32
#define HIST_B (hist_size*hist_size)
#define HIST_S 200
#define MERG_W 256

__global__ void CollectHistograms(float *hists_d, uint *bins_d, float *wimd_d, uint dataCount)
{
  __shared__ float hist_s[HIST_W*HIST_B];
  int lx = threadIdx.x;
  float *base_s = hist_s + lx;
  #pragma unroll
  for (int i=0;i<HIST_B;i++)
    hist_s[lx + i*HIST_W] = 0;
  __syncthreads();
  uint step = (__umul24(blockDim.x, gridDim.x))*4;
  uint pos = (__umul24(blockIdx.x, blockDim.x) + lx)*4;
  uint4 *bins4_d = (uint4*)bins_d;
  float4 *wimd4_d = (float4*)wimd_d;
  for (;pos<dataCount;pos+=step) {
    uint4 bins = bins4_d[pos/4];
    float4 weig = wimd4_d[pos/4];
    base_s[__umul24(bins.x,HIST_W)] += weig.x;
    base_s[__umul24(bins.y,HIST_W)] += weig.y;
    base_s[__umul24(bins.z,HIST_W)] += weig.z;
    base_s[__umul24(bins.w,HIST_W)] += weig.w;
  } 
  __syncthreads();
  const uint mask = (HIST_W - 1);
  for (int j=0;j<(HIST_B+HIST_W-1)/HIST_W;j++) {
    if (lx<HIST_B) {
      float *base_s = hist_s + __umul24(lx, HIST_W);
      float sum = 0.0f;
      uint pos = lx & 15;
      #pragma unroll
      for (uint i=0;i<HIST_W;i+=4,pos+=4) 
	sum += base_s[(pos+0)&mask] + base_s[(pos+1)&mask] + base_s[(pos+2)&mask] + base_s[(pos+3)&mask];
      hists_d[blockIdx.x*HIST_B + lx] = sum;
    }
    lx += HIST_W;
  }
}

__global__ void MergeHistograms(float *hist_d, float *hists_d, uint histCount) 
{
  __shared__ float data[MERG_W];
  int lx = threadIdx.x;
  float sum = 0.0f;
  for (uint i=lx;i<histCount;i+=MERG_W)
    sum += hists_d[blockIdx.x + i*HIST_B];
  data[lx] = sum;
  for (uint stride = MERG_W/2;stride>0;stride>>=1) {
    __syncthreads();
    if (lx<stride)
      data[lx] += data[lx + stride];
  }
  if (lx==0)
    hist_d[blockIdx.x] = data[0];
}

#endif

__global__ void SumUpGradientsD(uchar *image_d, int *sums_d, int w, int h)
{
  __shared__ uchar rows[(SUG_W+16)*2];
  __shared__ int sums[SUG_W];
  int lx = threadIdx.x;
  int bx = blockIdx.x*blockDim.x;   
  int x = bx + lx;
  int y = (blockIdx.y*blockDim.y + threadIdx.y)*SUG_H;
  uchar *row1 = &rows[0];
  uchar *row2 = &rows[SUG_W+16];
  int bp = y*w + bx;
  int x4 = (bx + 4*lx);
  if (lx<SUG_W/4+1 && x4<w-3) { 
    int *irow = (int*)&image_d[bp];
    ((int*)row1)[lx] = irow[lx];
  }
  bp += w;
  __syncthreads();  
  int sumdiff2 = 0;
  for (int dy=1;dy<=SUG_H && (y+dy)<h;dy++) {
    if (lx<SUG_W/4+1 && x4<w-3) {
      int *irow = (int*)&image_d[bp]; 
      ((int*)row2)[lx] = irow[lx];
    }
    __syncthreads();
    if (x<w-1) {
      int val = row1[lx];
      int diff = val - row2[lx];
      sumdiff2 += diff*diff;
      diff = val - row1[lx+1]; 
      sumdiff2 += diff*diff;
    }
    bp += w;
    uchar *trow = row1;
    row1 = row2;
    row2 = trow;
    __syncthreads();
  }
  sums[lx] = sumdiff2;
  __syncthreads();
  int step = SUG_W/2;
  while (step>0) {
    if (lx<step)
      sums[lx] += sums[lx+step];
    step >>= 1;
    __syncthreads();
  }
  if (lx==0) 
    sums_d[blockIdx.y*gridDim.x + blockIdx.x] = sums[lx];
}

__global__ void SetGradientCostsD(uchar *image_d, float *costh_d, float *costv_d, float beta, float gamma, int w, int h)
{
  __shared__ uchar rows[(SGC_W+16)*(SGC_H+1)];
  const int lw = SGC_W + 16;
  int lx = threadIdx.x;
  int ly = threadIdx.y;
  int bx = blockIdx.x*blockDim.x;  
  int x = bx + lx;
  int y = blockIdx.y*blockDim.y + ly;
  int rp = y*w + bx;
  int p = rp + lx;
  int lp = ly*lw + lx;
  int x4 = (bx + 4*lx);
  if (lx<SGC_W/4+1 && x4<w-3 && y<h) {
    int *irow = (int*)&image_d[rp];
    int *lrow = (int*)&rows[ly*lw];
    lrow[lx] = irow[lx];
  }
  if (lx<SGC_W/4+1 && x4<w-3 && ly==0 && (y+SGC_H)<h) {
    int *irow = (int*)&image_d[rp + SGC_H*w];
    int *lrow = (int*)&rows[SGC_H*lw];
    lrow[lx] = irow[lx];
  } 
  __syncthreads();
  if (x<w && y<h) {
    int v0 = rows[lp];
    int vx = (x<w-1 ? rows[lp+1] : v0);
    int vy = (y<h-1 ? rows[lp+lw] : v0); 
    int dx = vx - v0;
    int dy = vy - v0;
    costh_d[p] = gamma*(0.1f + __expf(beta*dx*dx)); //%%%%
    costv_d[p] = gamma*(0.1f + __expf(beta*dy*dy)); //%%%%
  }
}

//======================================================================//

CudaSegment::CudaSegment(int w, int h) : 
  width(w), height(h)
{
  dim3 grid((width+SUG_W-1)/SUG_W, (height+SUG_H-1)/SUG_H);
  safeCall(hipMalloc((void**)&sums_d, sizeof(int)*grid.x*grid.y));
  safeCall(hipMalloc(&cimd_d, 3*width*height));
  safeCall(hipMalloc(&dimd_d, sizeof(float)*width*height));
  safeCall(hipMalloc(&vimd_d, width*height));
  safeCall(hipMalloc(&bins_d, sizeof(int)*width*height));
  int sz = sizeof(float)*width*height;
  int n = max_figures+2;
  safeCall(hipMalloc(&allocated_d, (3*n+2)*sz));
  costh_d = (float*)&allocated_d[0*sz];
  costv_d = (float*)&allocated_d[1*sz];
  for (int i=0;i<n;i++) {
    priors_d[i] = (float*)&allocated_d[(0*n+i+2)*sz];
    beliefs_d[i] = (float*)&allocated_d[(1*n+i+2)*sz];
    probs_d[i] = (float*)&allocated_d[(2*n+i+2)*sz];
  }
}

CudaSegment::~CudaSegment()
{
  safeCall(hipFree(sums_d));
  safeCall(hipFree(cimd_d));
  safeCall(hipFree(dimd_d));
  safeCall(hipFree(bins_d));
  safeCall(hipFree(vimd_d));
  safeCall(hipFree(allocated_d));
}

void CudaSegment::CopyProbsToDevice(Image<float> **probs_h)
{
  safeCall(hipMemcpy(probs_d[0], probs_h[0]->GetData(), sizeof(float)*width*height, hipMemcpyHostToDevice));
  safeCall(hipMemcpy(probs_d[1], probs_h[1]->GetData(), sizeof(float)*width*height, hipMemcpyHostToDevice));
  safeCall(hipMemcpy(probs_d[2], probs_h[2]->GetData(), sizeof(float)*width*height, hipMemcpyHostToDevice));
}

#if 0  // Only possible if histograms are small

void CudaSegment::CreateHistograms(FgBgSegment &segment)
{ 
  float *hist_d, *hists_d;
  uint count = width*height;
  const int blockSize = HIST_W*HIST_S;
  uint numBlocks = (count+blockSize-1)/blockSize;
  safeCall(hipMalloc((void **)&hist_d, HIST_B*sizeof(float)));
  safeCall(hipMalloc((void **)&hists_d, numBlocks*HIST_B*sizeof(float)));
  for (int j=0;j<3;j++) {
    CollectHistograms<<<numBlocks, HIST_W>>>(hists_d, bins_d, probs_d[j], count);
    checkMsg("Kernel execution failed");
    safeCall(hipDeviceSynchronize());
    MergeHistograms<<<HIST_B, MERG_W>>>(hist_d, hists_d, numBlocks);
    checkMsg("Kernel execution failed");
    safeCall(hipDeviceSynchronize());
    float hist_h[HIST_B];
    safeCall(hipMemcpy(hist_h, hist_d, HIST_B*sizeof(float), hipMemcpyDeviceToHost));
    float fac_old = 1.0f / (1.0f + FgBgSegment::ColorModel::weight);
    float fac_new = 1.0f - fac_old;
    float *hist = segment.ground.histogram;
    float *cost = segment.ground.colorcost;
    if (j==1) {
      hist = segment.surface.histogram;
      cost = segment.surface.colorcost;
    } else if (j==2) {
      hist = segment.figure.histogram;
      cost = segment.figure.colorcost;
    }
    float num = 0.0f;
    for (int i=0;i<HIST_B;i++) 
      num += hist_h[i];
    num = (num>0.0f ? num : 1e-6f);
    for (int i=0;i<HIST_B;i++) {
      hist[i] = fac_new*(hist_h[i]/num) + fac_old*hist[i];
      cost[i] = -2.0f*log(hist[i]);
    }
  }
  safeCall(hipFree(hist_d));
  safeCall(hipFree(hists_d));
}

#endif

float CudaSegment::ComputeBeta(uchar *image_d) 
{
  dim3 block(SUG_W, 1);
  dim3 grid((width+SUG_W-1)/SUG_W, (height+SUG_H-1)/SUG_H);
  SumUpGradientsD<<<grid, block>>>(image_d, sums_d, width, height);
  checkMsg("Kernel execution failed"); 
  safeCall(hipDeviceSynchronize());
  int *sums = new int[grid.x*grid.y];
  safeCall(hipMemcpy(sums, sums_d, sizeof(int)*grid.x*grid.y, hipMemcpyDeviceToHost));
  int sum = 0;
  for (int i=0;i<grid.x*grid.y;i++) 
    sum += sums[i];
  float avggrad = (float)sum/(width-1)/(height-1)/2;
  float beta = -1.0f/(2.0f*avggrad);
  delete [] sums;
  return beta;
}

void CudaSegment::SetGradientCosts(uchar *image_d, float beta, float gamma)
{
  dim3 block(SGC_W, SGC_H);
  dim3 grid((width+SGC_W-1)/SGC_W, (height+SGC_H-1)/SGC_H);
  SetGradientCostsD<<<grid, block>>>(image_d, costh_d, costv_d, beta, gamma, width, height); 
  checkMsg("Kernel execution failed"); 
  safeCall(hipDeviceSynchronize());
}
  
void CudaSegment::Execute(FgBgSegment &segment, 
			  Image<uchar> &cimg, 
			  Image<float> &dimg, int numFigures, bool download)
{
  TimerCPU timer0(2800);
  if (download) {
  
    //  cimg.StoreRGB("/u/jbohg/GPUCimg.pgm");
    //    dimg.Store("/u/jbohg/GPUDimg.pgm");
 
    // Upload RGB image and disparity data 
    safeCall(hipMemcpy(cimd_d, cimg.GetData(), 3*width*height, hipMemcpyHostToDevice));
    safeCall(hipMemcpy(dimd_d, dimg.GetData(), sizeof(float)*width*height, hipMemcpyHostToDevice));
    // Convert from RGB to HSV
    dim3 block1(HSV_W, HSV_H);
    dim3 grid1((width+HSV_W-1)/HSV_W, (height+HSV_H-1)/HSV_H);
    TimerCPU timerc(2800);
    ::RGBToHSV<<<grid1, block1>>>(cimd_d, bins_d, vimd_d, width, height); 
    checkMsg("Kernel execution failed");
    safeCall(hipDeviceSynchronize()); 
    float delayc = timerc.read();
  }  

  // Set prior probability constants 
  const int num = max_figures + 2;
  float constd[num] = { 2.0f*log((float)segment.drange/2.0f), log(segment.surface.spread_d) };
  float constp[num] = { 2.0f*log((float)width*height), 2.0f*log((float)width*height) };
  float const0[num] = { -2.0*log(0.45f), -2.0*log(0.45f) };
  float constu[num] = { -2.0*log(0.40f), -2.0*log(0.40f) };
  for (int f=0;f<numFigures;f++) {
    constd[f+2] = log(segment.figures[f]->spread3d(2,2)); 
    constp[f+2] = log(segment.figures[f]->spread3d.determinant()) - constd[f+2]; 
    const0[f+2] = -2.0*log(0.10f);
    constu[f+2] = -2.0*log(0.20f);
  }
  if (segment.surface.spread_d>2.0f && false) {  //%%%%
    const0[0] = -2.0*log(0.90f);                 //%%%% Ignore surface if it's too 'thick'
    const0[1] = -2.0*log(1e-12f);                //%%%%
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(constd_d), constd, sizeof(float)*num));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(constp_d), constp, sizeof(float)*num));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(const0_d), const0, sizeof(float)*num));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(constu_d), constu, sizeof(float)*num));
  
  // Copy model parameters
  float sconstants[] = { segment.surface.alpha, segment.surface.beta, segment.surface.disp, 1.0f/segment.surface.spread_d };
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(sconstants_d), sconstants, sizeof(float)*4));
  float fispread3d_h[max_figures*9];
  float fposition3d_h[max_figures*3];
  for (int f=0;f<numFigures;f++) {
    for (int i=0;i<3;i++) 
      fposition3d_h[f*3+i] = segment.figures[f]->position3d(i);
    Matrix3 ivar = segment.figures[f]->spread3d;
    if (ivar.determinant()!=0.0)
      ivar = ivar.invert();
    for (int j=0;j<3;j++) 
      for (int i=0;i<3;i++)  
	fispread3d_h[f*9+j*3+i] = ivar(j, i);
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(fposition3d_d), fposition3d_h, sizeof(fposition3d_h)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(fispread3d_d), fispread3d_h, sizeof(fispread3d_h)));
  
  // Create colour histograms 
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(scolcost_d), segment.surface.colorcost, sizeof(float)*hist_size*hist_size));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(gcolcost_d), segment.ground.colorcost, sizeof(float)*hist_size*hist_size));
  float fcolcost_h[max_figures*hist_size*hist_size];
  for (int f=0;f<numFigures;f++) 
    memcpy(&fcolcost_h[f*hist_size*hist_size], segment.figures[f]->colorcost, sizeof(float)*hist_size*hist_size);
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(fcolcost_d), fcolcost_h, sizeof(float)*hist_size*hist_size*numFigures));
 
  // Compute prior probabilities     
  dim3 block2(PRI_W, PRI_H); 
  dim3 grid2((width+PRI_W-1)/PRI_W, (height+PRI_H-1)/PRI_H);
  ComputePriors<<<grid2, block2>>>(bins_d, dimd_d, segment.drange, priors_d[0], priors_d[1], priors_d[2], numFigures, width, height);
  checkMsg("Kernel execution failed");
  safeCall(hipDeviceSynchronize());
#if 0
  Image<float> dumpimg(width, height);    
  safeCall(hipMemcpy(dumpimg.GetData(), priors_d[0], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dumpimg.Store("prior0.pgm", true, false);
  safeCall(hipMemcpy(dumpimg.GetData(), priors_d[1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dumpimg.Store("prior1.pgm", true, false);
  safeCall(hipMemcpy(dumpimg.GetData(), priors_d[2], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dumpimg.Store("prior2.pgm", true, false);
  safeCall(hipMemcpy(dumpimg.GetData(), priors_d[3], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dumpimg.Store("prior3.pgm", true, false);
#endif 
    
  // Compute beta factor and set gradient costs
  float beta = ComputeBeta(vimd_d);
  SetGradientCosts(vimd_d, beta, segment.gradWeight); 
 
  // Perform belief propagation 
  int loops = 5;
  int depth = 4; 
  if (numFigures==0) { 
    CudaBelief<2> cudabelief(width, height, false);
    cudabelief.Execute(priors_d, beliefs_d, costh_d, costv_d, loops, depth);
  } else if (numFigures==1) { 
    CudaBelief<3> cudabelief(width, height, false);
    cudabelief.Execute(priors_d, beliefs_d, costh_d, costv_d, loops, depth);
  } else if (numFigures==2) {  
    CudaBelief<4> cudabelief(width, height, false);
    cudabelief.Execute(priors_d, beliefs_d, costh_d, costv_d, loops, depth);
  } else if (numFigures==3) {
    CudaBelief<5> cudabelief(width, height, false);
    cudabelief.Execute(priors_d, beliefs_d, costh_d, costv_d, loops, depth);
  } else if (numFigures==4) { 
    CudaBelief<6> cudabelief(width, height, false);
    cudabelief.Execute(priors_d, beliefs_d, costh_d, costv_d, loops, depth);
  } else if (numFigures==5) {
    CudaBelief<7> cudabelief(width, height, false);
    cudabelief.Execute(priors_d, beliefs_d, costh_d, costv_d, loops, depth); 
  } else if (numFigures==6) {
    CudaBelief<8> cudabelief(width, height, false);
    cudabelief.Execute(priors_d, beliefs_d, costh_d, costv_d, loops, depth);
  }

  // Convert from beliefs to probabilities
  dim3 block3(BTP_W, BTP_H); 
  dim3 grid3((width+BTP_W-1)/BTP_W, (height+BTP_H-1)/BTP_H);
  BeliefsToProbs<<<grid3, block3>>>(beliefs_d[0], beliefs_d[1], beliefs_d[2], probs_d[0], probs_d[1], probs_d[2], numFigures, width, height);
  checkMsg("Kernel execution failed");
  safeCall(hipDeviceSynchronize()); 

  // Copy back probabilities to host
  safeCall(hipMemcpy(segment.ground.probabilities.GetData(),  probs_d[0], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  safeCall(hipMemcpy(segment.surface.probabilities.GetData(), probs_d[1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  for (int f=0;f<numFigures;f++) 
    safeCall(hipMemcpy(segment.figures[f]->probabilities.GetData(),  probs_d[f+2], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  safeCall(hipDeviceSynchronize());
  float delayt = timer0.read();
  std::cout << "CudaSegment      Time: " << delayt << " ms " << width*height/delayt/1e3 << " Mpixels per second" << std::endl;
}

 
 
