#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011, Mårten Björkman (celle@csc.kth.se) 
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *
 *  1.Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  2.Redistributions in binary form must reproduce the above
 *    copyright notice, this list of conditions and the following
 *    disclaimer in the documentation and/or other materials provided
 *    with the distribution.  
 *  3.The name of Mårten Björkman may not be used to endorse or
 *    promote products derived from this software without specific
 *    prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include <hip/hip_math_constants.h>
#include "cudautils.h"
#include "timercpu.h" 

#define CBE_W 32  // ComputeBeliefs
#define CBE_H 16
#define CUD_W 32  // CollectUp & CollectDown
#define CUD_H 16
#define SDP_W 16  // ScaleDownPriors
#define SDP_H 16

__global__ void ComputeBeliefsD(float *prior, float *msgl, float *msgr, float *msgu, float *msgd, float *belief, int w, int h) 
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int p = y*w + x;
  if (x<w && y<h) {
    belief[p] = prior[p] + msgl[p] + msgr[p] + msgu[p] + msgd[p];
  }   
}

template<int dim>
__global__ void CollectLeft(float *costh, float *beliefs, float *msgs, float *msgos, int w, int h)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int p = y*w + x;
  int sz = w*h;
  float msg[dim];
  if (x<w-1 && y<h) {
    float minh = HIP_MAX_NORMAL_F;
    for (int f=0;f<dim;f++) {
      msg[f] = beliefs[f*sz+p] - msgs[f*sz+p];
      minh = min(msg[f], minh);
    }
    float minc = minh + costh[p]; 
    for (int f=0;f<dim;f++) 
      msgos[f*sz+p+1] = min(msg[f], minc) - minh;
  }
}

template<int dim>
__global__ void CollectRight(float *costh, float *beliefs, float *msgs, float *msgos, int w, int h)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int p = y*w + x;
  int sz = w*h;
  float msg[dim];
  if (x>0 && x<w && y<h) {
    float minh = HIP_MAX_NORMAL_F;
    for (int f=0;f<dim;f++) {
      msg[f] = beliefs[f*sz+p] - msgs[f*sz+p];
      minh = min(msg[f], minh);
    }
    float minc = minh + costh[p-1]; 
    for (int f=0;f<dim;f++) 
      msgos[f*sz+p-1] = min(msg[f], minc) - minh;
  }
}

template<int dim>
__global__ void CollectUp(float *costv, float *beliefs, float *msgs, float *msgos, int w, int h)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int p = y*w + x;
  int p0 = p - w;
  int sz = w*h;
  float msg[dim];
  if (x<w && y<h && y>0) {
    float minh = HIP_MAX_NORMAL_F;
    for (int f=0;f<dim;f++) {
      msg[f] = beliefs[f*sz+p0] - msgs[f*sz+p0];
      minh = min(msg[f], minh);
    }
    float minc = minh + costv[p0]; 
    for (int f=0;f<dim;f++) 
      msgos[f*sz+p] = min(msg[f], minc) - minh;
  }
}

template<int dim>
__global__ void CollectDown(float *costv, float *beliefs, float *msgs, float *msgos, int w, int h)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int p = y*w + x;
  int p0 = p + w;
  int sz = w*h;
  float msg[dim];
  if (x<w && y<h-1) {
    float minh = HIP_MAX_NORMAL_F;
    for (int f=0;f<dim;f++) {
      msg[f] = beliefs[f*sz+p0] - msgs[f*sz+p0];
      minh = min(msg[f], minh);
    }
    float minc = minh + costv[p0]; 
    for (int f=0;f<dim;f++) 
      msgos[f*sz+p] = min(msg[f], minc) - minh;
  }
} 

__global__ void ScaleDownPriorsD(float *priors1, int width1, int height1, float *priors2, int width2, int height2, int dim)
{
  int x2 = blockIdx.x*blockDim.x + threadIdx.x;
  int y2 = blockIdx.y*blockDim.y + threadIdx.y;
  int d = y2/height2;
  int x1 = 2*x2;
  int y1 = 2*y2 + (height1 - 2*height2)*d;
  float *ptr2 = priors2 + (x2 + y2*width2);
  float *ptr1 = priors1 + (x1 + y1*width1);
  if (x2<width2 && d<dim)
    ptr2[0] = ptr1[0] + ptr1[1] + ptr1[width1] + ptr1[width1+1];
}
 
__global__ void ScaleDownGradCostsD(float *costh1, float *costv1, int width1, float *costh2, float *costv2, int width2, int height2)
{
  int x2 = blockIdx.x*blockDim.x + threadIdx.x;
  int y2 = blockIdx.y*blockDim.y + threadIdx.y;
  int x1 = 2*x2;
  int y1 = 2*y2;
  int p2 = x2 + y2*width2;
  int p1 = x1 + y1*width1;
  float *ptrh1 = costh1 + p1;
  float *ptrv1 = costv1 + p1;
  if (x2<width2 && y2<height2) {
    costh2[p2] = ptrh1[1] + ptrh1[width1+1];
    costv2[p2] = ptrv1[width1] + ptrv1[width1+1];
  }
}
 
__global__ void ScaleUpMessagesD(float *msgs1, int width1, int height1, float *msgs2, int width2, int height2, int dim)
{
  int x2 = blockIdx.x*blockDim.x + threadIdx.x;
  int y2 = blockIdx.y*blockDim.y + threadIdx.y;
  int d = y2/height2;
  int x1 = 2*x2;
  int y1 = 2*y2 + (height1 - 2*height2)*d;
  float *ptr2 = msgs2 + (x2 + y2*width2);
  float *ptr1 = msgs1 + (x1 + y1*width1);
  if (x2<width2 && d<dim)
    ptr1[0] = ptr1[1] = ptr1[width1] = ptr1[width1+1] = ptr2[0];
}
 
//====================================================================================//

template<int dim>
CudaBelief<dim>::CudaBelief(int w, int h, bool copyFromHost_) : width(w), height(h), copyFromHost(copyFromHost_)
{
  int sz = sizeof(float)*w*h;
  int totsz = 8*dim*sz;
  safeCall(hipMalloc((void**)&allocated_d, totsz)); 
  for (int i=0;i<2*dim;i++) {
    msgl_d[i] = (float*)&allocated_d[(0*dim+i)*sz];
    msgr_d[i] = (float*)&allocated_d[(2*dim+i)*sz];
    msgu_d[i] = (float*)&allocated_d[(4*dim+i)*sz];
    msgd_d[i] = (float*)&allocated_d[(6*dim+i)*sz];
  }
  if (copyFromHost) {
    for (int i=0;i<dim;i++) {
      safeCall(hipMalloc((void**)&prior_d[i], sz));
      safeCall(hipMalloc((void**)&belief_d[i], sz));
    }
  }
}

template<int dim>
CudaBelief<dim>::~CudaBelief()
{
  safeCall(hipFree(allocated_d));
  if (copyFromHost) {
    for (int i=0;i<dim;i++) {
      safeCall(hipFree(prior_d[i]));
      safeCall(hipFree(belief_d[i]));
    }
  }  
}

template<int dim>
void CudaBelief<dim>::ComputeBeliefs(int loop)
{
  int s = dim*(loop&1);
  dim3 block(CBE_W, CBE_H);
  dim3 grid((width+CBE_W-1)/CBE_W, (height+CBE_H-1)/CBE_H);
  for (int i=0;i<dim;i++) 
    ComputeBeliefsD<<<grid, block>>>(prior_d[i], msgl_d[s+i], msgr_d[s+i], msgu_d[s+i], 
      msgd_d[s+i], belief_d[i], width, height);
  checkMsg("Kernel execution failed"); 
  safeCall(hipDeviceSynchronize());
#if 0
  std::cout << "ComputeBeliefs" << std::endl;
  Image<float> dump(width, height);
  safeCall(hipMemcpy(dump.GetData(),  prior_d[1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dump.Store("dump5.pgm", true, false);
  safeCall(hipMemcpy(dump.GetData(),  msgr_d[s+1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dump.Store("dump2.pgm", true, false);
  safeCall(hipMemcpy(dump.GetData(),  msgu_d[s+1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dump.Store("dump3.pgm", true, false);
  safeCall(hipMemcpy(dump.GetData(),  msgd_d[s+1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dump.Store("dump4.pgm", true, false);
  safeCall(hipMemcpy(dump.GetData(),  msgl_d[s+1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dump.Store("dump1.pgm", true, false);
  safeCall(hipMemcpy(dump.GetData(),  belief_d[1], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  dump.Store("dump6.pgm", true, false);
#endif
}

template<int dim>
void CudaBelief<dim>::CollectMessages(float *costh_d, float *costv_d, int loop)
{
  int s1 = dim*(loop&1);
  int s2 = s1^dim;
  dim3 blocku(CUD_W, CUD_H);
  dim3 gridu((width+CUD_W-1)/CUD_W, (height+CUD_H-1)/CUD_H);
  CollectLeft<dim><<<gridu, blocku>>>(costh_d, belief_d[0], msgr_d[s1], msgl_d[s2], width, height); 
  CollectRight<dim><<<gridu, blocku>>>(costh_d, belief_d[0], msgl_d[s1], msgr_d[s2], width, height); 
  CollectUp<dim><<<gridu, blocku>>>(costv_d, belief_d[0], msgd_d[s1], msgu_d[s2], width, height); 
  CollectDown<dim><<<gridu, blocku>>>(costv_d, belief_d[0], msgu_d[s1], msgd_d[s2], width, height); 
  checkMsg("Kernel execution failed"); 
  safeCall(hipDeviceSynchronize());
}

template<int dim>
void CudaBelief<dim>::Execute(float **priors, float **beliefs, float *costh_d, float *costv_d)
{
  if (copyFromHost) {
    TimerCPU timer0(2800);
    for (int i=0;i<dim;i++) 
      safeCall(hipMemcpy(prior_d[i], priors[i], sizeof(float)*width*height, hipMemcpyHostToDevice));
    float delay0 = timer0.read();
    std::cout << "MemoryCopy       Time: " << delay0 << " ms " << width*height/delay0/1e3 << " Mpixels per second" << std::endl;
  } else {
    for (int i=0;i<dim;i++) {  
      prior_d[i] = priors[i]; 
      belief_d[i] = beliefs[i];
    }
  }
  safeCall(hipMemset(msgl_d[0], 0, sizeof(float)*width*height*dim));
  safeCall(hipMemset(msgr_d[0], 0, sizeof(float)*width*height*dim));
  safeCall(hipMemset(msgu_d[0], 0, sizeof(float)*width*height*dim));
  safeCall(hipMemset(msgd_d[0], 0, sizeof(float)*width*height*dim));
  float delayt = 0.0f;
  int loops = 10; 
  for (int i=0;i<loops;i++) {
    TimerCPU timer1(2800);
    ComputeBeliefs(i);
    float delay1 = timer1.read();
    delayt += delay1;
    //std::cout << "ComputeBeliefs   Time: " << delay1 << " ms " << width*height/delay1/1e3 << " Mpixels per second" << std::endl;
    TimerCPU timer2(2800);
    CollectMessages(costh_d, costv_d, i);
    float delay2 = timer2.read();
    delayt += delay2;
    //std::cout << "CollectMessages  Time: " << delay2 << " ms " << width*height/delay2/1e3 << " Mpixels per second" << std::endl;
  }
  TimerCPU timer3(2800);  
  ComputeBeliefs(loops);
  if (copyFromHost) 
    for (int i=0;i<dim;i++) 
      safeCall(hipMemcpy(beliefs[i], belief_d[i], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  float delay3 = timer3.read();
  delayt += delay3;
  //std::cout << "ComputeBeliefs   Time: " << delay3 << " ms " << width*height/delay3/1e3 << " Mpixels per second" << std::endl;
  std::cout << "CudaBelief       Time: " << delayt << " ms " << width*height/delayt/1e3 << " Mpixels per second" << std::endl;
}

#include "pyra/tpimage.h" //%%%%

template<int dim>
void CudaBelief<dim>::Execute(float **priors, float **beliefs, float *costh_d, float *costv_d, int loops, int depth)
{
  if (copyFromHost) {
    TimerCPU timer0(2800);
    for (int i=0;i<dim;i++) 
      safeCall(hipMemcpy(prior_d[i], priors[i], sizeof(float)*width*height, hipMemcpyHostToDevice));
    float delay0 = timer0.read();
    std::cout << "MemoryCopy       Time: " << delay0 << " ms " << width*height/delay0/1e3 << " Mpixels per second" << std::endl;
  } else {
    for (int i=0;i<dim;i++) {  
      prior_d[i] = priors[i]; 
      belief_d[i] = beliefs[i];
    }
  }
  float delayt = 0.0f;
  if (depth>0) {
    TimerCPU timer4(2800);
    int width2 = width/2;
    int height2 = height/2;
    int sz = sizeof(float)*width2*height2;
    unsigned char *allocated2_d;
    float *prior2_d[dim], *belief2_d[dim];
    safeCall(hipMalloc(&allocated2_d, (2*dim+2)*sz));
    float *costh2_d = (float*)&allocated2_d[0*sz];
    float *costv2_d = (float*)&allocated2_d[1*sz];
    for (int i=0;i<dim;i++) {
      prior2_d[i] = (float*)&allocated2_d[(0*dim+i+2)*sz];
      belief2_d[i] = (float*)&allocated2_d[(1*dim+i+2)*sz];
    }    
    dim3 block1(SDP_W, SDP_H);
    dim3 grid1((width2+SDP_W-1)/SDP_W, (height2*dim+SDP_H-1)/SDP_H);
    ScaleDownPriorsD<<<grid1, block1>>>(prior_d[0], width, height, prior2_d[0], width2, height2, dim);
    checkMsg("Kernel execution failed");  // NOTE: assumes priors to be stored sequentially
    safeCall(hipDeviceSynchronize());
    dim3 block2(SDP_W, SDP_H);
    dim3 grid2((width2+SDP_W-1)/SDP_W, (height2+SDP_H-1)/SDP_H);
    ScaleDownGradCostsD<<<grid2, block2>>>(costh_d, costv_d, width, costh2_d, costv2_d, width2, height2);
    checkMsg("Kernel execution failed");  
    safeCall(hipDeviceSynchronize());
    CudaBelief<dim> cudabelief(width2, height2, false);  
    float delay4 = timer4.read();
    delayt += delay4; 
    cudabelief.Execute(prior2_d, belief2_d, costh2_d, costv2_d, loops, depth-1);
    TimerCPU timer5(2800);
    safeCall(hipFree(allocated2_d));
    float *mptrs1[4] = { msgl_d[0], msgr_d[0], msgu_d[0], msgd_d[0] };
    float *mptrs2[4] = { cudabelief.msgl_d[0], cudabelief.msgr_d[0], cudabelief.msgu_d[0], cudabelief.msgd_d[0] };
    for (int i=0;i<4;i++) {
      ScaleUpMessagesD<<<grid1, block1>>>(mptrs1[i], width, height, mptrs2[i], width2, height2, dim);
      checkMsg("Kernel execution failed");  
      safeCall(hipDeviceSynchronize());
    }
    float delay5 = timer5.read();
    delayt += delay5; 
  } else {
    safeCall(hipMemset(msgl_d[0], 0, sizeof(float)*width*height*dim));
    safeCall(hipMemset(msgr_d[0], 0, sizeof(float)*width*height*dim));
    safeCall(hipMemset(msgu_d[0], 0, sizeof(float)*width*height*dim));
    safeCall(hipMemset(msgd_d[0], 0, sizeof(float)*width*height*dim));
  }
  int l = loops; //(depth>0 ? loops : height/2);
  for (int i=0;i<l;i++) {
    TimerCPU timer1(2800);
    ComputeBeliefs(i);
    float delay1 = timer1.read();
    delayt += delay1;
    //std::cout << "ComputeBeliefs   Time: " << delay1 << " ms " << width*height/delay1/1e3 << " Mpixels per second" << std::endl;
    TimerCPU timer2(2800);
    CollectMessages(costh_d, costv_d, i);
    float delay2 = timer2.read();
    delayt += delay2;
    //std::cout << "CollectMessages  Time: " << delay2 << " ms " << width*height/delay2/1e3 << " Mpixels per second" << std::endl;
  }
  TimerCPU timer3(2800);  
  ComputeBeliefs(l);
  if (copyFromHost)  
    for (int i=0;i<dim;i++) 
      safeCall(hipMemcpy(beliefs[i], belief_d[i], sizeof(float)*width*height, hipMemcpyDeviceToHost));
  float delay3 = timer3.read();
  delayt += delay3; 

  //std::cout << "CudaBelief       Time: " << delayt << " ms " << width*height/delayt/1e3 << " Mpixels per second" << std::endl;
}
  
